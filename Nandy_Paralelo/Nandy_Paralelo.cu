#include "hip/hip_runtime.h"
#include <stdio.h> // biblioteca input/output padr�o do C 
#include <stdlib.h>
#include <time.h>
#include <malloc.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>

#define nThreadsPorBloco 16

#include "C:\Users\Comum\Documents\Unicamp\Mestrado\Programa��o\_bibliotecas\Estruturas.h"
#include "C:\Users\Comum\Documents\Unicamp\Mestrado\Programa��o\_bibliotecas\Algebra_Linear_serial.h"
#include "C:\Users\Comum\Documents\Unicamp\Mestrado\Programa��o\_bibliotecas\Algebra_Linear_paralelo.h"
#include "C:\Users\Comum\Documents\Unicamp\Mestrado\Programa��o\_bibliotecas\Auxiliares_serial.h"
#include "C:\Users\Comum\Documents\Unicamp\Mestrado\Programa��o\_bibliotecas\Auxiliares_paralelo.h"
#include "C:\Users\Comum\Documents\Unicamp\Mestrado\Programa��o\_bibliotecas\GA_Paralelo.h"


//---------------------------------------------------------------------------------------
int main (void) {
//---------------------------------------------------------------------------------------
	
	clock_t			time_i = 999, time_f = 999
					,	time_pgm_i = 999, time_pgm_f = 999;
	
	// Obtem o momento inicial
	time_pgm_i = clock();
		
	struct parametros				parametrosGA, *devParametrosGA;
	struct parametros_Metodo	parametrosMetodo, *devParametrosMetodo;
	struct generation				geracao[2], *devGeracao0, *devGeracao1;
	unsigned int					iGeracao, qtdeElementos;

	cudaErrorCheck(hipMalloc((void **)&devGeracao0, sizeof(struct generation)));
	cudaErrorCheck(hipMalloc((void **)&devGeracao1, sizeof(struct generation)));

	inicializa_Parametros(&parametrosGA, &parametrosMetodo);
	inicializa_Semente();

	// imprime o cabe�alho
	//imprimeTempo(0, 0, 0 , 0, 1, 2, time_i, time_f);
	
	// -----------------------------------------------------------------------
	// Copia para a GPU os par�metros do GA e do M�todo.
	// -----------------------------------------------------------------------
	cudaErrorCheck(hipMalloc((void **)&devParametrosGA, sizeof(struct parametros)));
	cudaErrorCheck(hipMemcpy(devParametrosGA, &parametrosGA, sizeof(struct parametros), hipMemcpyHostToDevice));

	cudaErrorCheck(hipMalloc((void **)&devParametrosMetodo, sizeof(struct parametros_Metodo)));
	cudaErrorCheck(hipMemcpy(devParametrosMetodo, &parametrosMetodo, sizeof(struct parametros_Metodo), hipMemcpyHostToDevice));

	// -----------------------------------------------------------------------
	// HAMILTONIANO
	// -----------------------------------------------------------------------
	time_i = clock();
	float *hostHamiltoniano, *devHamiltoniano;
	hostHamiltoniano = (float *)malloc(parametrosGA.numGenes * parametrosGA.numGenes*sizeof(float));
	cudaErrorCheck(hipMalloc((void **)&devHamiltoniano, parametrosGA.numGenes * parametrosGA.numGenes * sizeof(float)));
	unsigned int numBlocosLinha, numBlocosColuna;
	numBlocosLinha = (parametrosGA.numGenes + nThreadsPorBloco)/nThreadsPorBloco;
	numBlocosColuna = (parametrosGA.numGenes + nThreadsPorBloco)/nThreadsPorBloco;
	dim3 ThreadsH(nThreadsPorBloco, nThreadsPorBloco);
	dim3 BlocosH(numBlocosLinha, numBlocosColuna);
	dev_Gera_Matriz_de_Coope_naGPU<<<BlocosH,ThreadsH>>>(devHamiltoniano, devParametrosGA);
	time_f = clock();
	//imprimeTempo(1, 0, 1 , 0, 1, 2, time_i, time_f);
	//testaHamiltoniano(hostHamiltoniano, devHamiltoniano, &parametrosGA);

	// no link abaixo h� informa��es sobre a aparente n�o execu��o da fun��o acima
	// http://stackoverflow.com/questions/21982826/cuda-kernel-function-not-called
	// http://stackoverflow.com/questions/11587227/basic-cuda-getting-kernels-to-run-on-the-device-using-c
	
	// medindo os erros:
	// http://devblogs.nvidia.com/parallelforall/how-query-device-properties-and-handle-errors-cuda-cc/	
	
	
	// -----------------------------------------------------------------------
	// N�MEROS PSEUDO-ALEAT�RIOS
	// -----------------------------------------------------------------------
	// semente para os n�meros aleat�rios
	unsigned int hostSemente, *devSemente;
	hostSemente = (unsigned int)time(NULL);
	cudaErrorCheck(hipMalloc((void **)&devSemente, sizeof(unsigned int)));
	cudaErrorCheck(hipMemcpy(devSemente, &hostSemente, sizeof(unsigned int), hipMemcpyHostToDevice));
	
	// Inicia estado para n�meros pseudoaleatr�rios.
	hiprandState *dev_estadosCURAND;
	unsigned int numEstadosTotal = parametrosGA.numIndividuos*parametrosGA.numGenes;
	cudaErrorCheck(hipMalloc((void **)&dev_estadosCURAND, numEstadosTotal*sizeof(hiprandState)));

	unsigned int numBlocosCuRand = (numEstadosTotal + nThreadsPorBloco)/nThreadsPorBloco;

	dim3 numThreadsCURAND(nThreadsPorBloco);
	dim3 numBlocosCURAND(numBlocosCuRand);

	devInicializaCURAND<<<numBlocosCURAND,numThreadsCURAND>>>(	devGeracao0,
																					devSemente,
																					dev_estadosCURAND,
																					devParametrosGA);
	cudaErrorCheck(hipFree(devSemente));
	
	// -----------------------------------------------------------------------
	// POPULACAO INICIAL
	// -----------------------------------------------------------------------
	time_i = clock();
	GeraPopulacaoInicial_paralela(devGeracao0, &parametrosGA, devParametrosGA, dev_estadosCURAND);
	time_f = clock();
	//imprimeTempo(1, 0, 1, 0, 2, 2, time_i, time_f);
	//testePopulacaoInicial(&geracao[0], devGeracao0, &parametrosGA);
	
	// -----------------------------------------------------------------------
	// MATRIZ IDENTIDADE
	// -----------------------------------------------------------------------
	time_i = clock();
	float *devMatrizIdentidade;
	qtdeElementos = parametrosGA.numGenes * parametrosGA.numGenes;
	cudaErrorCheck(hipMalloc((void **)&devMatrizIdentidade, qtdeElementos*sizeof(float)));
	numBlocosLinha = (parametrosGA.numGenes + nThreadsPorBloco)/nThreadsPorBloco;
	numBlocosColuna = (parametrosGA.numGenes + nThreadsPorBloco)/nThreadsPorBloco;
	dim3 ThreadsI(nThreadsPorBloco,nThreadsPorBloco);
	dim3 BlocosI(numBlocosLinha,numBlocosColuna);
	dev_GeraMatrizIdentidade<<<BlocosI,ThreadsI>>>(devMatrizIdentidade, devParametrosGA);
	time_f = clock();
	imprimeTempo(1, 0, 1 , 0, 3, 2, time_i, time_f);
	//testeMatrizIdentidade_GPU(devMatrizIdentidade,&parametrosGA);

	// -----------------------------------------------------------------------
	// ITERA��O PRINCIPAL
	// -----------------------------------------------------------------------

	// - teste selecao - pode ser removido depois
	struct testeGeracao_s *dev_testeGeracao_s;
	cudaErrorCheck(hipMalloc((void **)&dev_testeGeracao_s, sizeof(testeGeracao_s)));
	//
	imprimeComportamentoFitness_GPU(0, 0, 0, 0, 0, devGeracao0, devParametrosMetodo);

	for (iGeracao = 0; iGeracao < parametrosGA.numGeracoes; iGeracao++) {
		
		// -----------------------------------------------------------------------
		// FITNESS -- INICIO
		// -----------------------------------------------------------------------

		time_i = clock();	
		Fitness(	'2',
					devHamiltoniano,
					devGeracao0,
					&parametrosGA,
					devParametrosGA,
					devParametrosMetodo,
					devMatrizIdentidade);

		//teste_Fitness_GPU(devGeracao0, &parametrosGA);
		time_f = clock();
		//imprimeTempo(1, 0, 1, iGeracao, 8, 2, time_i, time_f);
		imprimeComportamentoFitness_GPU(1, 0, 1, 2, iGeracao, devGeracao0, devParametrosMetodo);


		// -----------------------------------------------------------------------
		// SELECAO -- INICIO
		// -----------------------------------------------------------------------

		time_i = clock();
		//testeSelecao(0, devGeracao0, &parametrosGA, dev_testeGeracao_s);
		
		Selecao(	devGeracao0,
					devGeracao1,
					&parametrosGA,
					devParametrosGA,
					dev_estadosCURAND,
					dev_testeGeracao_s);
		
		//testeSelecao(1, devGeracao1, &parametrosGA, dev_testeGeracao_s);
		time_f = clock();		
		//imprimeTempo(1, 0, 1, iGeracao, 5, 2, time_i, time_f);
		
		// -----------------------------------------------------------------------
		// CROSSOVER - INICIO
		// -----------------------------------------------------------------------

		time_i = clock();
		//testeCrossOver(2, devGeracao1, &parametrosGA, dev_testeGeracao_s);
		//testeCrossOver(0, devGeracao1, &parametrosGA, dev_testeGeracao_s);
		CrossOver1Ponto(
			devGeracao1,
			devGeracao0,
			&parametrosGA,
			devParametrosGA,
			dev_estadosCURAND,
			dev_testeGeracao_s);
		//testeCrossOver(1, devGeracao0, &parametrosGA, dev_testeGeracao_s);
		time_f = clock();
		//imprimeTempo(1, 0, 1, iGeracao, 6, 2, time_i, time_f);

		// -----------------------------------------------------------------------
		// MUTACAO - INICIO
		// -----------------------------------------------------------------------

		time_i = clock();
		//testeMutacao(0, devGeracao0, &parametrosGA);
		Mutacao(
			devGeracao0,
			&parametrosGA,
			devParametrosGA,
			dev_estadosCURAND);
		//testeMutacao(1, devGeracao0, &parametrosGA);
		time_f = clock();
		//imprimeTempo(1, 0, 1, iGeracao, 7, 2, time_i, time_f);		
				
	} // ===> Fim da itera��o principal
						
	
	time_i = clock();
	Fitness(	'2',
				devHamiltoniano,
				devGeracao0,
				&parametrosGA,
				devParametrosGA,
				devParametrosMetodo,
				devMatrizIdentidade);

	time_f = clock();
	//imprimeTempo(1, 0, 1, iGeracao, 8, 2, time_i, time_f);
	imprimeComportamentoFitness_GPU(1, 0, 1, 2, iGeracao, devGeracao0, devParametrosMetodo);


	// Obtem o momento final em segundos e imprime a dura��o total do programa
	time_pgm_f = clock();
	imprimeTempo(1, 0, 1, iGeracao, 0, 2, time_pgm_i, time_pgm_f);

	// Retorno dos valores da GPU. Aqui
	printf("\n\n");
	printf("--------------------------------------------------------\n");
	printf("Geracao %d final\n", iGeracao);
	printf("--------------------------------------------------------\n");

	cudaErrorCheck(hipMemcpy(&geracao[0], devGeracao0, sizeof(struct generation), hipMemcpyDeviceToHost) );
	imprimeGeracao(&geracao[0], &parametrosGA);

	hostHamiltoniano = NULL; free(hostHamiltoniano);
	cudaErrorCheck(hipFree(devHamiltoniano));
	cudaErrorCheck(hipFree(devGeracao0));
	cudaErrorCheck(hipFree(devGeracao1));
	cudaErrorCheck(hipFree(devParametrosGA));
	cudaErrorCheck(hipFree(devParametrosMetodo));
	cudaErrorCheck(hipFree(devMatrizIdentidade));
	//cudaErrorCheck(hipFree(dev_testeGeracao_s));

	return 0;
}